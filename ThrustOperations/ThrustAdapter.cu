#include "hip/hip_runtime.h"
//
// Created by hkumar on 31.08.20.
//

#include "ThrustAdapter.cuh"

ThrustAdapter::ThrustAdapter(ThrustCompute *ATC_obj) {
    ATC = ATC_obj;
}

vector<int> ThrustAdapter::selection(vector<int> data, string operation, int value) {
    thrust::device_vector<int> deviceData = ATC->getThrustGpuData(data);
    thrust::device_vector<int> deviceResult(data.size());

    vector<int> result;
    vector<int> durations;

    for (int i = 0; i <= 100; i++) {
        auto start = high_resolution_clock::now();  // start time

        // operation here
        deviceResult = ATC->thrustSelection(deviceData,operation,value);

        auto stop = high_resolution_clock::now(); // stop time
        auto duration = duration_cast<microseconds>(stop - start); // time taken for performing the operation
        if(i>0) {
            durations.push_back(
                    duration.count());  // since the initial load time is high, calculation is started from 2nd iteration
        }
        else{
            result = ATC->getThrustCpuData(deviceResult);
        }
    }


    std::cout << "Time taken for selection operation " << operation << "_" <<to_string(value)
              << " is " <<std::accumulate(durations.begin(),
                                          durations.end(), 0) / durations.size() << " microseconds" << std::endl;

    thrust::device_vector<int> buffer = result;
    thrust::exclusive_scan(buffer.begin(),buffer.end(),buffer.begin());

//    vector<int> host_buffer = ATC->getThrustCpuData(buffer);
//    int size = host_buffer.back();

    return result;
}

vector<int> ThrustAdapter::conjunction(vector<int> lhs, vector<int> rhs) {
    thrust::device_vector<int> deviceLHS = ATC->getThrustGpuData(lhs);
    thrust::device_vector<int> deviceRHS = ATC->getThrustGpuData(rhs);
    thrust::device_vector<int> deviceResult(lhs.size());

    vector<int> result;
    vector<int> durations;


    for (int i = 0; i <= 100; i++) {
        auto start = high_resolution_clock::now();  // start time

        // operation here
        deviceResult = ATC->thrustConjunction(deviceLHS,deviceRHS);

        auto stop = high_resolution_clock::now(); // stop time
        auto duration = duration_cast<microseconds>(stop - start); // time taken for performing the operation
        if(i>0) {
            durations.push_back(duration.count());  // since the initial load time is high, calculation is started from 2nd iteration
        }else{
            result = ATC->getThrustCpuData(deviceResult);
        }
    }

    std::cout << "Time taken for conjunction operation is " <<std::accumulate(durations.begin(),
                                                                              durations.end(), 0) / durations.size() << " microseconds" << std::endl;
    return result;
}

vector<int> ThrustAdapter::product(vector<int> lhs, vector<int> rhs) {
    thrust::device_vector<int> deviceLHS = ATC->getThrustGpuData(lhs);
    thrust::device_vector<int> deviceRHS = ATC->getThrustGpuData(rhs);
    thrust::device_vector<int> deviceResult(lhs.size());

    vector<int> result;
    vector<int> durations;


    for (int i = 0; i <= 100; i++) {
        auto start = high_resolution_clock::now();  // start time

        // operation here
        deviceResult = ATC->thrustProduct(deviceLHS,deviceRHS);

        auto stop = high_resolution_clock::now(); // stop time
        auto duration = duration_cast<microseconds>(stop - start); // time taken for performing the operation
        if(i>0) {
            durations.push_back(duration.count());  // since the initial load time is high, calculation is started from 2nd iteration
        }else{
            result = ATC->getThrustCpuData(deviceResult);
        }
    }

    std::cout << "Time taken for product operation is " <<std::accumulate(durations.begin(),
                                                                          durations.end(), 0) / durations.size() << " microseconds" << std::endl;
    return result;
}

int ThrustAdapter::sum(vector<int> data) {

    thrust::device_vector<int> deviceData = ATC->getThrustGpuData(data);
    int result;

    vector<int> durations;

    for (int i = 0; i <= 100; i++) {
        auto start = high_resolution_clock::now();  // start time

        // operation here
        result = ATC->thrustSum(deviceData);

        auto stop = high_resolution_clock::now(); // stop time
        auto duration = duration_cast<microseconds>(stop - start); // time taken for performing the operation
        if(i>0) {
            durations.push_back(duration.count());  // since the initial load time is high, calculation is started from 2nd iteration
        }
    }

    std::cout << "Time taken for sum operation is " <<std::accumulate(durations.begin(),
                                                                      durations.end(),
                                                                      0) / durations.size() << " microseconds" << std::endl;
    return result;
}

vector<int> ThrustAdapter::sort(vector<int> data, int order) {

    thrust::device_vector<int> deviceData = ATC->getThrustGpuData(data);

    thrust::device_vector<int> deviceResult(data.size());

    vector<int> result;
    vector<int> durations;

    for (int i = 0; i <= 100; i++) {
        auto start = high_resolution_clock::now();  // start time

        // operation here
        deviceResult = ATC->thrustSort(deviceData,order);

        auto stop = high_resolution_clock::now(); // stop time
        auto duration = duration_cast<microseconds>(stop - start); // time taken for performing the operation
        if(i>0) {
            durations.push_back(duration.count());  // since the initial load time is high, calculation is started from 2nd iteration
        }else{
            result = ATC->getThrustCpuData(deviceResult);
        }
    }


    std::cout << "Time taken for sort operation is " <<std::accumulate(durations.begin(),
                                                                       durations.end(), 0) / durations.size() << " microseconds" << std::endl;

    return result;
}

float ThrustAdapter::avg(vector<int> data) {

    thrust::device_vector<int> deviceData = ATC->getThrustGpuData(data);
    float result;

    vector<int> durations;

    for (int i = 0; i <= 100; i++) {
        auto start = high_resolution_clock::now();  // start time

        // operation here
        result = ATC->thrustAvg(deviceData);

        auto stop = high_resolution_clock::now(); // stop time
        auto duration = duration_cast<microseconds>(stop - start); // time taken for performing the operation
        if(i>0) {
            durations.push_back(duration.count());  // since the initial load time is high, calculation is started from 2nd iteration
        }
    }

    std::cout << "Time taken for average operation is " <<std::accumulate(durations.begin(),
                                                                          durations.end(),
                                                                          0) / durations.size() << " microseconds" << std::endl;
    return result;
}

int ThrustAdapter::countIf(vector<int> data) {

    thrust::device_vector<int> deviceData = ATC->getThrustGpuData(data);
    int result;

    vector<int> durations;

    for (int i = 0; i <= 100; i++) {
        auto start = high_resolution_clock::now();  // start time

        // operation here
        result = ATC->thrustCountIf(deviceData);

        auto stop = high_resolution_clock::now(); // stop time
        auto duration = duration_cast<microseconds>(stop - start); // time taken for performing the operation
        if(i>0) {
            durations.push_back(duration.count());  // since the initial load time is high, calculation is started from 2nd iteration
        }
    }

    std::cout << "Time taken for count operation is " <<std::accumulate(durations.begin(),
                                                                        durations.end(),
                                                                        0) / durations.size() << " microseconds" << std::endl;

    return result;
}

int ThrustAdapter::count(vector<int> data) {

    thrust::device_vector<int> deviceData = ATC->getThrustGpuData(data);
    int result;

    vector<int> durations;

    for (int i = 0; i <= 100; i++) {
        auto start = high_resolution_clock::now();  // start time

        // operation here
        result = ATC->thrustCount(deviceData);

        auto stop = high_resolution_clock::now(); // stop time
        auto duration = duration_cast<microseconds>(stop - start); // time taken for performing the operation
        if(i>0) {
            durations.push_back(duration.count());  // since the initial load time is high, calculation is started from 2nd iteration
        }
    }

    std::cout << "Time taken for count operation is " <<std::accumulate(durations.begin(),
                                                                        durations.end(),
                                                                        0) / durations.size() << " microseconds" << std::endl;
    return result;
}

void ThrustAdapter::join2(vector<int> parent, vector<int> child){
    int offset = pow(2,16);

//        do Block Nested Loop Join
    int begin_R = 0, begin_S = 0;
    int end_R = begin_R + offset, end_S = begin_S + offset;

    cout << "parent size: " << parent.size() << endl;
    cout << "child size: " << child.size() << endl;

    int r_size = parent.size();
    int s_size = child.size();

    int r[r_size];
    int s[s_size];

    std::copy(parent.begin(),parent.end(),r);
    std::copy(child.begin(),child.end(),s);
    parent.clear();
    child.clear();

    int sum = 0;
    vector<int> temp_R(offset);
    vector<int> temp_S(offset);
    vector<int> durations;
    vector<int> result;
    vector<vector<int>> joinResult;

    for (int i = 0; i < 1; i++) {
        auto start = high_resolution_clock::now();  // start time

        thrust::device_vector<int> deviceResult(offset); //*offset

        for (auto&& i: parent | sliced(0, r_size/offset)){
            for(auto&& j: child | sliced(0, s_size/offset)){

                std::copy(r+begin_R,r+end_R,temp_R.begin());
                std::copy(s+begin_S,s+end_S,temp_S.begin());

                thrust::device_vector<int> deviceLHS = ATC->getThrustGpuData(temp_R);
                thrust::device_vector<int> deviceRHS = ATC->getThrustGpuData(temp_S);
//                thrust::device_vector<int> deviceResult(temp_R.size() * temp_S.size());

//                cout << "R: ( " << begin_R << " , " << end_R << ") and "
//                                                                "S: ( " << begin_S << " , " << end_S << endl;

                deviceResult = ATC->thrustJoin(deviceLHS,deviceRHS);

                result = ATC->getThrustCpuData(deviceResult);
//                sum += thrust::count(deviceResult.begin(),deviceResult.end(),1);

                vector<vector<int>> tempJoin = joinTuples(temp_R,temp_S,offset,result,i,j,offset);

                joinResult.insert(joinResult.end(),tempJoin.begin(),tempJoin.end());

                begin_S += offset;
                end_S += offset;
            }
            cout << "R: ( " << begin_R << " , " << end_R << " ) " << endl;
//            cout << "Sum: " << sum << endl;
            begin_S = 0;
            end_S = begin_S + offset;
            begin_R += offset;
            end_R += offset;
        }

        auto stop = high_resolution_clock::now(); // stop time
        auto duration = duration_cast<microseconds>(stop - start); // time taken for performing the operation
        cout << "Time taken: " << duration.count() << endl;
        if(i>0) {
            durations.push_back(duration.count());  // since the initial load time is high, calculation is started from 2nd iteration
        }
    }

    cout << "Total number of joins: " << sum;
}

vector<int> ThrustAdapter::join(vector<int> parent, vector<int> child) {
    thrust::device_vector<int> deviceLHS = ATC->getThrustGpuData(parent);
    thrust::device_vector<int> deviceRHS = ATC->getThrustGpuData(child);
//    cout<<parent.size() <<" : "<<child.size()<<" : "<<parent.size() * child.size()*sizeof(int)<<" : total allocated size"<<endl;
//    thrust::device_vector<int> deviceResult(parent.size() * child.size());

    thrust::device_vector<int> deviceResult(child.size());

    vector<int> result;
    vector<int> durations;

    int R = 0;
    int S = 0;
    int offset = 50;



//    for(auto it = deviceLHS.begin()+R; it != deviceLHS.begin()+R+offset; it++){
//        for(auto it2 = deviceRHS.begin()+S; it2 != deviceRHS.begin()+S+offset;it2++){
//
//            thrust::device_reference<int> valueLHS = *it;
//            thrust::device_reference<int> valueRHS = *it;
//
//            S = S+ offset;
//        }
//        R = R+ offset;
//    }


    for (int i = 0; i <= 100; i++) {
        auto start = high_resolution_clock::now();  // start time

        // operation here
        deviceResult = ATC->thrustJoin(deviceLHS,deviceRHS);

        auto stop = high_resolution_clock::now(); // stop time
        auto duration = duration_cast<microseconds>(stop - start); // time taken for performing the operation
        if(i>0) {
            durations.push_back(duration.count());  // since the initial load time is high, calculation is started from 2nd iteration
        }else{
            result = ATC->getThrustCpuData(deviceResult);
        }
    }

    std::cout << "Time taken for join operation is " <<std::accumulate(durations.begin(),
                                                                       durations.end(), 0) / durations.size() << " microseconds" << std::endl;
    return result;
}

vector<int> ThrustAdapter::prefixSum(vector<int> data) {
    thrust::device_vector<int> deviceData = ATC->getThrustGpuData(data);
    int count = ATC->thrustCountIf(deviceData);
    thrust::device_vector<int> deviceResult(count);

    vector<int> durations;
    vector<int> result;

    for (int i = 0; i <= 100; i++) {
        auto start = high_resolution_clock::now();  // start time

        // operation here
        deviceResult = ATC->thrustPrefixSum(deviceData);

        auto stop = high_resolution_clock::now(); // stop time
        auto duration = duration_cast<microseconds>(stop - start); // time taken for performing the operation
        if(i>0) {
            durations.push_back(duration.count());  // since the initial load time is high, calculation is started from 2nd iteration
        }else{
            result = ATC->getThrustCpuData(deviceResult);
        }
    }

    std::cout << "Time taken for prefix sum operation is " <<std::accumulate(durations.begin(),
                                                                             durations.end(),
                                                                             0) / durations.size() << " microseconds" << std::endl;

    return result;
}

int ThrustAdapter::findMax(vector<int> data) {

    thrust::device_vector<int> deviceData = ATC->getThrustGpuData(data);
    int result;

    vector<int> durations;

    for (int i = 0; i <= 100; i++) {
        auto start = high_resolution_clock::now();  // start time

        // operation here
        result = ATC->thrustFindMax(deviceData);

        auto stop = high_resolution_clock::now(); // stop time
        auto duration = duration_cast<microseconds>(stop - start); // time taken for performing the operation
        if(i>0) {
            durations.push_back(duration.count());  // since the initial load time is high, calculation is started from 2nd iteration
        }
    }

    std::cout << "Time taken for findMax operation is " <<std::accumulate(durations.begin(),
                                                                          durations.end(),
                                                                          0) / durations.size() << " microseconds" << std::endl;
    return result;
}

int ThrustAdapter::findMin(vector<int> data) {

    thrust::device_vector<int> deviceData = ATC->getThrustGpuData(data);
    int result;

    vector<int> durations;

    for (int i = 0; i <= 100; i++) {
        auto start = high_resolution_clock::now();  // start time

        // operation here
        result = ATC->thrustFindMin(deviceData);

        auto stop = high_resolution_clock::now(); // stop time
        auto duration = duration_cast<microseconds>(stop - start); // time taken for performing the operation
        if(i>0) {
            durations.push_back(duration.count());  // since the initial load time is high, calculation is started from 2nd iteration
        }
    }

    std::cout << "Time taken for findMin operation is " <<std::accumulate(durations.begin(),
                                                                          durations.end(),
                                                                          0) / durations.size() << " microseconds" << std::endl;
    return result;
}