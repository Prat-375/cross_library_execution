#include "hip/hip_runtime.h"
//
// Created by hkumar on 31.08.20.
//

#pragma once
#include "afComputeOps.cuh"

af::array afCompute::getAFGpuData(std::vector<int> data) {
    int* hostData = &data[0];
    af::array deviceData((dim_t)data.size(), hostData, afHost);
    return deviceData;
}

vector<int> afCompute::getAFCpuData(af::array deviceData) {
    vector<int> hostData(deviceData.elements());
    deviceData.host(hostData.data());
    return hostData;
}

af::array afCompute::afSelection(af::array deviceData, string operation, int value) {

    af::array device_result;

    if(!operation.compare("GE")) {
        device_result = af::where(af::operator>=(deviceData, value));
    }
    else if(!operation.compare("LE")) {

        device_result = af::where(af::operator<=(deviceData, value));
    }
    else if(!operation.compare("G")) {
        device_result = af::where(af::operator>(deviceData, value));
    }
    else if(!operation.compare("L")) {
        device_result = af::where(af::operator<(deviceData, value));
    }
    else if(!operation.compare("EQ")) {
        device_result = af::where(af::operator==(deviceData, value));
    }
    else{
        device_result = af::where(af::operator!=(deviceData, value));
    }

    return device_result;
}

af::array afCompute::afConjunction(af::array deviceLHS, af::array deviceRHS) {
    af::array result = af::setIntersect(deviceLHS,deviceRHS,true);
    return result;
}

af::array afCompute::afProduct(af::array deviceLHS, af::array deviceRHS) {
    af::array result = af::operator*(deviceLHS,deviceRHS);
    return result;
}

int afCompute::afSum(af::array deviceData) {
//    af::array device_result = af::sum<int>(data,(dim_t)0);
//    int result = device_result.row(0).elements();
    int result = af::sum<int>(deviceData,(dim_t)0);
    return result;
}

float afCompute::afAvg(af::array deviceData) {
    float result = af::mean<float>(deviceData);
    return result;
}

int afCompute::afCountIf(af::array deviceData) {
    return af::count<int>(deviceData);
}

int afCompute::afCount(af::array deviceData) {
    return deviceData.elements();
}

af::array afCompute::afJoin(af::array parent, af::array child) {

    af::array A = af::constant(0, parent.elements(), child.elements());
    gfor(af::seq i,parent.elements()){
        for(int j=0;j<child.elements();j++){
            af::array condition = (child(j) == parent(i));
            A(i,j) =  (!condition).as(f32) * 0 + (condition).as(f32) * 1;
        }
    }
    return A;
}

// Please note arrayfire does not need prefix sum operation as it already returns the indices in selection operation
// the function has been added below for consistency to run all libraries in sequence
af::array afCompute::afPrefixSum(af::array deviceSelData) {
    return deviceSelData;
}

int afCompute::afFindMax(af::array deviceData) {
    return af::max<int>(deviceData);
}

int afCompute::afFindMin(af::array deviceData) {
    return af::min<int>(deviceData);
}