#include "hip/hip_runtime.h"
//
// Created by hkumar on 31.08.20.
//

#pragma once
#include "afComputeOps.cuh"

af::array afCompute::getAFGpuData(std::vector<int> data) {
    int* hostData = &data[0];
    af::array deviceData((dim_t)data.size(), hostData, afHost);
    return deviceData;
}

af::array afCompute::getAFGpuData(std::vector<float> data) {
    float * hostData = &data[0];
    af::array deviceData((dim_t)data.size(), hostData, afHost);
    return deviceData;
}

vector<int> afCompute::getAFCpuData(af::array deviceData) {
    vector<int> hostData(deviceData.elements());
    deviceData.host(hostData.data());
    return hostData;
}

vector<float> afCompute::getAFFloatCpuData(af::array deviceData) {
    vector<float> hostData(deviceData.elements());
    deviceData.host(hostData.data());
    return hostData;
}

af::array afCompute::afSelection(af::array deviceData, string operation, int value) {

    af::array device_result;

    if(!operation.compare("GE")) {
        device_result = af::where(af::operator>=(deviceData, value));
    }
    else if(!operation.compare("LE")) {

        device_result = af::where(af::operator<=(deviceData, value));
    }
    else if(!operation.compare("G")) {
        device_result = af::where(af::operator>(deviceData, value));
    }
    else if(!operation.compare("L")) {
        device_result = af::where(af::operator<(deviceData, value));
    }
    else if(!operation.compare("EQ")) {
        device_result = af::where(af::operator==(deviceData, value));
    }
    else{
        device_result = af::where(af::operator!=(deviceData, value));
    }

    return device_result;
}

af::array afCompute::afSelectionArrays(af::array lhs, string operation, af::array rhs) {

    af::array device_result;

    if(!operation.compare("GE")) {
        device_result = af::where(af::operator>=(lhs, rhs));
    }
    else if(!operation.compare("LE")) {

        device_result = af::where(af::operator<=(lhs, rhs));
    }
    else if(!operation.compare("G")) {
        device_result = af::where(af::operator>(lhs, rhs));
    }
    else if(!operation.compare("L")) {
        device_result = af::where(af::operator<(lhs, rhs));
    }
    else if(!operation.compare("EQ")) {
        device_result = af::where(af::operator==(lhs, rhs));
    }
    else{
        device_result = af::where(af::operator!=(lhs, rhs));
    }

    return device_result;
}

af::array afCompute::afConjunction(af::array deviceLHS, af::array deviceRHS) {
    af::array result = af::setIntersect(deviceLHS,deviceRHS,true);
    return result;
}

af::array afCompute::afProduct(af::array deviceLHS, af::array deviceRHS) {
    af::array result = af::operator*(deviceLHS,deviceRHS);
    return result;
}

int afCompute::afSum(af::array deviceData) {
//    af::array device_result = af::sum<int>(data,(dim_t)0);
//    int result = device_result.row(0).elements();
    int result = af::sum<int>(deviceData,(dim_t)0);
    return result;
}

float afCompute::afAvg(af::array deviceData) {
    float sum = af::sum<float>(deviceData,(dim_t)0);
    float result = sum/deviceData.elements();
    return result;
}

int afCompute::afCountIf(af::array deviceData,int value) {
//        return af::count<int>(deviceData);
    af::array index = af::where(af::operator==(deviceData,value));
    return af::count<int>(index);
}

int afCompute::afCount(af::array deviceData) {
    return deviceData.elements();
}


af::array test_nested_loop(af::array::array_proxy result,af::array::array_proxy child,af::array parent) { //

    for(int j=0 ; j < parent.elements(); j++){
        result = af::select(child==parent(j),j,result);
    }
    return result;
}


//af::array nested_loop_join(af::array::array_proxy child,
//                           af::array parent,
//                           af::array::array_proxy result) {
//
//    for (int j = 0; j < parent.elements(); j++){
//        result = af::select(child==parent(j),j,result);
//    }
//    return result;
//}

af::array afCompute::afJoin(af::array parent, af::array child) {

// Why arrayfire join is slow:
// https://stackoverflow.com/questions/50242141/arrayfire-cuda-application-is-extremely-slow-in-the-first-minute
// https://github.com/arrayfire/arrayfire-python/issues/140

/*
 *          condition = (child(i) == parent(j));
            A(i) = (condition)*j + (!condition)*A(i);
 */
    af::array A = af::constant(-1,child.elements());

    gfor(af::seq i, child.elements()) { //
        test_nested_loop(A(i),child(i),parent);
    }
//
    A = A.as(af::dtype::s32);

    return A;
}

// Please note arrayfire does not need prefix sum operation as it already returns the indices in selection operation
// the function has been added below for consistency to run all libraries in sequence
af::array afCompute::afPrefixSum(af::array deviceSelData) {
    return deviceSelData;
}

int afCompute::afFindMax(af::array deviceData) {
    return af::max<int>(deviceData);
}

int afCompute::afFindMin(af::array deviceData) {
    return af::min<int>(deviceData);
}

af::array afCompute::afSort(af::array deviceData, int order) {

    af::array sorted_data;
    if(order){
        sorted_data = af::sort(deviceData,0, false);
    }else{
        sorted_data = af::sort(deviceData);
    }
    return sorted_data;
}

af::array afCompute::afGroupBy(af::array keys, af::array values) {
    af::array keys_out;
    af::array values_out;

    af::sumByKey(keys_out,values_out,keys,values);
    return values_out;
}

af::array afCompute::afCountByKey(af::array data){
    af::array keys_out;
    af::array values_out;

    af::array temp = data;

    af::sumByKey(keys_out,values_out,data,temp);
    values_out = af::operator/(values_out,keys_out);

    return values_out;
}

af::array afCompute::afSumOfVectors(af::array vec1, af::array vec2) {
    af::array result(vec1.elements());

    gfor(af::seq i, vec1.elements()){
        result(i) = vec1(i) + vec2(i);
    }

    result = result.as(af::dtype::s32);

    return result;
}

af::array afCompute::afSortByKey(af::array data, af::array dependent_data, int order) {

    af::array sorted_index;
    af::array sorted_value;
//
//    af::sort(sorted_value,sorted_index,data);
//    af::array result(dependent_data.elements());
//
//
//    gfor(af::seq i,dependent_data.elements()){
//        af::array pos = sorted_index(i);
//        result(i) = dependent_data(pos);
//    }
//
//    result = result.as(s32);
//
//    return result;
    if (order) {
        af::sort(sorted_index, sorted_value, data, dependent_data,0,false);
    }else{
        af::sort(sorted_index, sorted_value, data, dependent_data);
    }
    return sorted_value;
}

af::array afCompute::afAvgByKey(af::array keys, af::array values) {

    af::array sums = afGroupBy(keys,values);

    af::array counts = afCountByKey(keys);

    af::array average = af::operator/(sums,counts);

    return average;
}